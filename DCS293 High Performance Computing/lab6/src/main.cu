#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>

#include "../include/matmul.cuh"
#include "../include/helper.hpp"

#define testRounds 10
#define num_threads 4
// #define DEBUG

int main(int argc, char *argv[])
{
	init_rand();

	int m, n, k;
	m = n = k = strtol(argv[1], NULL, 10);
	printf("Conducting Multiplication of two matricies with size %d*%d\n", m, n);
	cudaMat *mata = init_mat(m, n, 1, colMajor),
			*matb = init_mat(n, k, 1, colMajor);

	// Likely useful for double-precision operations.
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	for(int round = 0;round < testRounds;round++) {
		printf("Testing Round %d:\n", round);

		// cudaMat *omp_matc = matmul_omp(mata, matb);
		cudaMat *cuda_matc = matmul_cuda(mata, matb);
		cudaMat *ompCuda_matc = matmul_cuda_multiDevices(mata, matb, num_threads);
		cudaMat *cublas_matc = matmul_cuBLAS(mata, matb, 1);

	#ifdef DEBUG
		// print_mat(omp_matc);
		print_mat(cuda_matc);
		print_mat(ompCuda_matc);
		print_mat(cublas_matc);
	#endif
		
		printf("Result check %s\n\n", equal_mat(cuda_matc, cublas_matc) ? "accepted" : "failed");
		
		// free_mat(omp_matc);
		free_mat(cuda_matc);
		free_mat(ompCuda_matc);
		free_mat(cublas_matc);
	}
	// printf("Average execution time: %lf\n", avgTime_omp/testRounds);
	printf("Average execution time on GPU: %lf\n", avgTime_cuda/testRounds);
	printf("Average execution time on %d GPUS: %lf\n", num_threads, avgTime_ompCuda/testRounds);
	printf("Average execution time of cuBLAS, on %d GPUS: %lf\n", num_threads, avgTime_cublas/testRounds);
	return 0;
}