#include <iostream>
using namespace std;
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#include "../include/conv.cuh"
#include "../include/helper.hpp"

#define testRounds 10
#define num_devices 4
#define kernelSize 3

int main(int argc, char *argv[])
{
    int imageSize = strtol(argv[1], NULL, 10);
    cudaImage image = cudaImage(imageSize, imageSize, 3, 1, rowMajor);
    cudaImage kernel = cudaImage(kernelSize, kernelSize, 3, 1, rowMajor);

#ifdef DEBUG
    // image.print();
    // kernel.print();
#endif

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    for(int stride = 1;stride <= 3;stride++) {
        cudaImage *omp_resImage = NULL, 
                  *cud_resImage = NULL,
                  *col_resImage = NULL,
                  *cudnn_resImage = NULL;
        
        omp_avgTime = cuda_avgTime = col_avgTime = cudnn_avgTime = 0;
        for(int i = 0;i < testRounds;i++) {
            omp_resImage = conv_omp(image, kernel, stride);
            cud_resImage = conv_cuda(image, kernel, stride);
            col_resImage = conv_im2col(image, kernel, stride);
            cudnn_resImage = conv_cudnn(image, kernel, stride);

            if(i == testRounds-1) {
            #ifdef DEBUG
                omp_resImage->print();
                cud_resImage->print();
                col_resImage->print();
                cudnn_resImage->print();
            #endif
                printf("Result check %s\n", (*cudnn_resImage) == (*cud_resImage) ? "accepted" : "failed");
            }

            if(omp_resImage) delete omp_resImage;
            if(cud_resImage) delete cud_resImage;
            if(col_resImage) delete col_resImage;
            if(cudnn_resImage) delete cudnn_resImage;
        }
        printf("Average time on %d CPU threads: %lf\n", omp_get_num_procs(), omp_avgTime/testRounds);
        printf("Average time on CUDA: %lf\n", cuda_avgTime/testRounds);
        printf("Average time on CUDA, im2col version: %lf\n", col_avgTime/testRounds);
        printf("Average time on cuDNN library: %lf\n", cudnn_avgTime/testRounds);
    }

    return 0;
}