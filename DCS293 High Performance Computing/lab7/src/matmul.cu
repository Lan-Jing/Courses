#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "../include/conv.cuh"
#include "../include/helper.hpp"

// vector-matrix multiplication. 1-d image is treated as matrix.
// This trick would be very likely useless on GPU
__constant__ dtype sd_A[16384];
cudaImage* vmm_cuda(cudaImage &mata, cudaImage &matb, int om, int on)
{
    int nCol = mata.size();
    if(nCol != matb.m)
        return NULL;

    cudaImage *matc = new cudaImage(om, on, 1, 0, rowMajor);

    dtype* d_A, *d_B, *d_C;
    hipMalloc(&d_A, mata.size() * dsize);
    hipMalloc(&d_B, matb.size() * dsize);
    hipMalloc(&d_C, matc->size() * dsize);
    hipMemcpy(d_A, mata.mat, mata.size()*dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matb.mat, matb.size()*dsize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(sd_A), d_A, mata.size()*dsize);

    dim3 blockDim(blockSize, 1),
         gridDim((matb.n+blockSize-1)/blockSize, 1);

    col_stime = omp_get_wtime();
    vmm_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, 
                                      matb.m, matb.n);
    hipDeviceSynchronize();
    col_avgTime += omp_get_wtime()-col_stime;

    hipMemcpy(matc->mat, d_C, matc->size()*dsize, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return matc;
}

__global__ void vmm_kernel(vtype d_A, vtype d_B, vtype d_C, 
                           int n, int k)
{
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx < k) {
        dtype res = 0;
        
        for(int i = 0, bIdx = idx;i < n;i++, bIdx += k)
            res += sd_A[i] * d_B[bIdx];

        d_C[idx] = res;
    }
}