#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "../include/conv.cuh"
#include "../include/helper.hpp"

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

__constant__ dtype sd_k[blockSize*blockSize];

// have the depth dimension change slowest.
int cudaImage::index(int i, int rIdx, int cIdx) const
{
    if(type == rowMajor) {
        return i*m*n + rIdx * n + cIdx;
    } else {
        return i*m*n + cIdx * m + rIdx;
    }
}

int cudaImage::size() const
{
    return m*n*d;
}

dtype& cudaImage::operator [] (int idx) const
{
    return mat[idx];
}

bool cudaImage::operator == (cudaImage &other) const
{
    if(m != other.m || n != other.n || d != other.d)
        return false;
    
    dtype maxError = 0;
    for(int i = 0;i < d;i++) {
        int rIdx;
    #pragma omp parallel for private(rIdx)
        for(rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int index = this->index(i, rIdx, cIdx);
                maxError = fmax(maxError, fabs(mat[index]-other.mat[index]));
            }
    }
    if(maxError >= eps) {
        printf("Maximum Error: %lf, required: %lf\n", maxError, eps);
        return false;
    } else {
        return true;
    }
}

void cudaImage::operator = (cudaImage &other) 
{
    if(mat != NULL) {
        delete [] mat;
    }

    m = other.m, n = other.n, d = other.d, type = other.type;
    
    int size = this->size();
    mat = new dtype[size];

    memcpy(mat, other.mat, size*dsize);
}

cudaImage::cudaImage()
{
    m = n = d = 0;
    mat = NULL;
}

cudaImage::cudaImage(int _m, int _n, int _d, int _mode, matType _type)
{
    m = _m, n = _n, d = _d, type = _type;
    
    long long int size = this->size();
    mat = new dtype[size];

    if(_mode == 1) {
        for(int i = 0;i < size;i++)
        #ifdef DEBUG
            mat[i] = 1;
        #else
            mat[i] = rand_float(RAND_RBOUND);
        #endif
    }
}

cudaImage::cudaImage(cudaImage &other, matType _type)
{
    m = other.m, n = other.n, d = other.d, type = _type;

    int size = this->size();
    mat = new dtype[size];

    if(type == other.type) {
        memcpy(mat, other.mat, size*dsize);
    } else {
        for(int i = 0;i < d;i++)
            for(int rIdx = 0;rIdx < m;rIdx++)
                for(int cIdx = 0;cIdx < n;cIdx++) {
                    int index = this->index(i, rIdx, cIdx);
                    mat[index] = other.mat[index];
                }
    }
}

cudaImage::~cudaImage()
{
    delete [] mat;
}

void cudaImage::print() const
{
    printf("Arranged in %s order\n", type ? "col-major" : "row-major");
    for(int i = 0;i < d;i++) {
        printf("Channel: %d\n", i);
        for(int rIdx = 0;rIdx < m;rIdx++) {
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int index = this->index(i, rIdx, cIdx);
                printf("%lf ", mat[index]); 
            }
            printf("\n");
        }
    }
    printf("\n");
}

// Rotate the index manner as well as the matrix itself.
void cudaImage::alter_major()
{
    matType newType = type == rowMajor ? colMajor : rowMajor;
    cudaImage tmpImage = cudaImage(*this, newType);
    for(int i = 0;i < d;i++)
        for(int rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int tIdx = tmpImage.index(i, rIdx, cIdx),
                    idx = this->index(i, rIdx, cIdx);
                tmpImage[tIdx] = (*this)[idx];
            }
    *this = tmpImage;
}

// return a padded image and corresponding output size.
cudaImage* cudaImage::padding(int &om, int &on, int km, int kn, int s) const
{
    int rpl, rpr, cpl, cpr;
    get_padding(rpl, rpr, cpl, cpr, om, on, m, n, km, kn, s);
    cudaImage *padImage = new cudaImage(m+rpl+rpr, n+cpl+cpr, d, 0, rowMajor);

#ifdef DEBUG
    printf("Padding Info: %d %d %d %d %d %d\n", rpl, rpr, cpl, cpr, om, on);
#endif

    memset(padImage->mat, 0, padImage->size()*dsize);
    for(int i = 0;i < d;i++) {
        int rIdx;
    #pragma omp parallel for private(rIdx)
        for(rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int prIdx = rIdx + rpl,
                    pcIdx = cIdx + cpl,
                    pIdx = padImage->index(i, prIdx, pcIdx),
                    idx = index(i, rIdx, cIdx);
               (*padImage)[pIdx] = (*this)[idx]; 
            }
    }
    return padImage;
}

cudaImage* cudaImage::im2col(int &om, int &on, int km, int kn, int s) const
{
    cudaImage *colImage = NULL, *padImage = NULL;
    if(km > m || kn > n)
        return colImage;
    
    padImage = padding(om, on, km, kn, s);
    // Be cautious. This operation is extermely mem-hungry.
    colImage = new cudaImage(km*kn*d, om*on, 1, 0, rowMajor);

    for(int i = 0;i < d;i++) {
        // for each row in the colImage
        int sIdx = 0;
    #pragma omp parallel for private(sIdx)
        for(sIdx = 0;sIdx < km*kn;sIdx++) {
            int sy = sIdx / km, sx = sIdx % km,
                cIdx = colImage->index(0, i*km*kn+sIdx, 0);
            for(int k = 0;k < om*on;k++, cIdx++) {
                int y = k / om, x = k % om,
                    pIdx = padImage->index(i, (sy+y*s), (sx+x*s));
                (*colImage)[cIdx] = (*padImage)[pIdx];
            }
        }
    }
    return colImage;
}

/*
============================== Implementation of convolution =====================
*/

cudaImage* conv_omp(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL, *padImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int km = kernel.m, kn = kernel.n, om, on;
    padImage = input.padding(om, on, km, kn, s);
    resImage = new cudaImage(om, on, 1, 0, rowMajor);

    omp_stime = omp_get_wtime();
    // loop over each output pixel. BE CAREFUL to the indexing!!!
    for(int i = 0;i < input.d;i++) {    
        int rIdx;
    #pragma omp parallel for private(rIdx)
        for(rIdx = 0;rIdx < om;rIdx++)
            for(int cIdx = 0;cIdx < on;cIdx++) {
                dtype &out = (*resImage)[resImage->index(0, rIdx, cIdx)];
                if(i == 0) out = 0;
                for(int ky = 0;ky < km;ky++)
                    for(int kx = 0;kx < kn;kx++) {
                        int irIdx = rIdx * s + ky,
                            icIdx = cIdx * s + kx,
                            kIdx = kernel.index(i, ky, kx),
                            iIdx = padImage->index(i, irIdx, icIdx);
                        out += (*padImage)[iIdx] * kernel[kIdx];
                    }
            }
    }
    omp_avgTime += omp_get_wtime() - omp_stime;

    delete padImage;
    return resImage;
}

cudaImage* conv_cuda(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL, *padImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int km = kernel.m, kn = kernel.n, om, on;
    padImage = input.padding(om, on, km, kn, s);
    resImage = new cudaImage(om, on, 1, 0, rowMajor);

    dtype *d_i, *d_o, *d_k;
    hipMalloc(&d_i, padImage->size() * dsize);
    hipMalloc(&d_o, resImage->size() * dsize);
    hipMalloc(&d_k, kernel.size() * dsize);
    hipMemcpy(d_i, padImage->mat, padImage->size() * dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_k, kernel.mat, kernel.size() * dsize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(sd_k), d_k, kernel.size() * dsize);

    dim3 blockDim(blockSize, blockSize),
         gridDim((om+blockSize-1)/blockSize, (on+blockSize-1)/blockSize);

    cuda_stime = omp_get_wtime();
    conv_kernel2<<<gridDim, blockDim>>>(d_i, d_k, d_o,
                                        padImage->m, padImage->n, s, padImage->d,
                                        km, kn,
                                        om, on);
    hipDeviceSynchronize();
    cuda_avgTime += omp_get_wtime() - cuda_stime;

    hipMemcpy(resImage->mat, d_o, resImage->size() * dsize, hipMemcpyDeviceToHost);
    hipFree(d_i);
    hipFree(d_o);
    hipFree(d_k);

    delete padImage;
    return resImage;
}

// No need to deal with the kernel, since it is already stored as a row vector.
cudaImage* conv_im2col(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL, *colImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int km = kernel.m, kn = kernel.n, om, on;
    colImage = input.im2col(om, on, km, kn, s);
    resImage = vmm_cuda(kernel, *colImage, om, on);

    delete colImage;
    return resImage;
}

cudaImage* conv_cudnn(cudaImage &input, cudaImage &kernel, int s) 
{
    cudaImage *resImage = NULL, *padImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int km = kernel.m, kn = kernel.n, kd = kernel.d, om, on;
    padImage = input.padding(om, on, km, kn, s);
    resImage = new cudaImage(om, on, 1, 0, rowMajor);

    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
    hipdnnDataType_t type = HIPDNN_DATA_FLOAT;

    hipdnnTensorDescriptor_t xDesc, yDesc;
    hipdnnCreateTensorDescriptor(&xDesc);
    hipdnnSetTensor4dDescriptor(xDesc, 
                               format, 
                               type, 
                               1, padImage->d, padImage->m, padImage->n);
    hipdnnCreateTensorDescriptor(&yDesc);
    hipdnnSetTensor4dDescriptor(yDesc, 
                               format, 
                               type, 
                               1, 1, om, on);
    
    hipdnnFilterDescriptor_t wDesc;
    hipdnnCreateFilterDescriptor(&wDesc);
    hipdnnSetFilter4dDescriptor(wDesc, 
                               type, 
                               format, 
                               1, kd, km, kn);

    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc, 
                                    0, 0, // already pad on the input
                                    s, s, // stride
                                    1, 1, // no dilation
                                    HIPDNN_CROSS_CORRELATION, 
                                    type);

#ifdef DEBUG
    // Check if descriptor for y is correctly set, given x, w and conv.
    int n, c, h, w;
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          xDesc,
                                          wDesc,
                                          &n, &c, &h, &w);
    printf("Specification vs. expected: N%d,%d C%d,%d H%d,%d W%d,%d\n", 1, n, 1, c, om, h, on, w);
#endif

    // Can be set via heuristic method.
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
    
    size_t workspaceSize = 0;
    void *workspace = NULL;
    hipdnnGetConvolutionForwardWorkspaceSize(handle, 
                                            xDesc, wDesc, convDesc, yDesc, 
                                            algo, 
                                            &workspaceSize);
    hipMalloc(&workspace, workspaceSize);

#ifdef DEBUG
    cout<<"Additional GPU mem needed:"<<workspaceSize<<endl;
#endif

    dtype *x, *k, *y;
    hipMalloc(&x, padImage->size()*dsize);
    hipMalloc(&y, resImage->size()*dsize);
    hipMalloc(&k, kernel.size()*dsize);
    hipMemcpy(x, padImage->mat, padImage->size()*dsize, hipMemcpyHostToDevice);
    hipMemcpy(k, kernel.mat, kernel.size()*dsize, hipMemcpyHostToDevice);

    cudnn_stime = omp_get_wtime();
    // Be cautious! Scalar values should be strictly set equal to other input. Float here. 
    const dtype alpha = 1.0, beta = 0.0;
    hipdnnConvolutionForward(handle,
                            &alpha, xDesc, x,
                            wDesc, k,
                            convDesc,
                            algo,
                            workspace,
                            workspaceSize,
                            &beta, yDesc, y);
    hipDeviceSynchronize();
    cudnn_avgTime += omp_get_wtime() - cudnn_stime;

    hipMemcpy(resImage->mat, y, resImage->size()*dsize, hipMemcpyDeviceToHost);
    hipFree(x);
    hipFree(y);
    hipFree(k);
    hipFree(workspace);

    hipdnnDestroyTensorDescriptor(xDesc);
    hipdnnDestroyTensorDescriptor(yDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyFilterDescriptor(wDesc);

    hipdnnDestroy(handle);
    delete padImage;
    return resImage;
}

// This kernel assumes all matrices arranged in row-major order.
__global__ void conv_kernel1(vtype d_i, vtype d_k, vtype d_o, 
                             const int im, const int in, const int s, const int d,
                             const int km, const int kn, 
                             const int om, const int on)
{
    int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
        cIdx = threadIdx.y + blockIdx.y * blockDim.y,
        rIdx_s = rIdx * s,
        cIdx_s = cIdx * s;

    dtype res = 0;
    if(rIdx < om && cIdx < on) {
        for(int i = 0;i < d;i++) {
            int irIdx = rIdx_s;
            for(int ky = 0;ky < km;ky++, irIdx++) {
                int icIdx = cIdx_s;
                for(int kx = 0;kx < kn;kx++, icIdx++) {
                    int kIdx = i*km*kn + ky*kn + kx,
                        iIdx = i*im*in + irIdx*in + icIdx;
                    res += d_i[iIdx] * d_k[kIdx];
                }
            }
        }
        d_o[rIdx * on + cIdx] = res;
    }
}

// input image -> shared memory
// the filter -> constant memory
__global__ void conv_kernel2(dtype *d_i, dtype *d_k, dtype *d_o, 
                             const int im, const int in, const int s, const int d,
                             const int km, const int kn,
                             const int om, const int on)
{
    // index in the thread block
    int brIdx = threadIdx.x, 
        bcIdx = threadIdx.y,
        brIdx_i = brIdx * s, 
        bcIdx_i = bcIdx * s,
    // index in the whole image
        grIdx = brIdx + blockIdx.x * blockDim.x, 
        gcIdx = bcIdx + blockIdx.y * blockDim.y,
        grIdx_i = grIdx * s, 
        gcIdx_i = gcIdx * s;

    dtype res = 0;
    __shared__ dtype sd_i[3*blockSize+10][3*blockSize+10];
    if(grIdx < om && gcIdx < on) {
        for(int i = 0;i < d;i++) {
            // each block loabds pixels it need into the shared mem, including the apron area.
            for(int r = 0;(brIdx == blockSize-1 || grIdx == om-1) ? r < km : r < s;r++)
                for(int c = 0;(bcIdx == blockSize-1 || gcIdx == on-1) ? c < kn : c < s;c++) {
                    int iIdx = i*im*in + (grIdx_i+r)*in + gcIdx_i+c;
                    sd_i[brIdx_i+r][bcIdx_i+c] = d_i[iIdx];
                }
            __syncthreads();
            int rIdx = brIdx_i;
            for(int ky = 0;ky < km;ky++, rIdx++) {
                int cIdx = bcIdx_i;
                for(int kx = 0;kx < kn;kx++, cIdx++) {
                    int kIdx = i*km*kn + ky*kn + kx;
                    res += sd_i[rIdx][cIdx] * sd_k[kIdx];
                }
            }
            __syncthreads();
        }
        d_o[grIdx * on + gcIdx] = res;
    }
}
