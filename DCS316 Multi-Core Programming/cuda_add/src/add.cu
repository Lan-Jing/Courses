#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <cstdio>
#include <cmath>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../include/add.hpp"
#include "../include/helper.hpp"

// a warper for checking cuda function calls
#define gpuCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// The matrix
matrix::matrix()
{
	height = width = 0;
}

matrix::matrix(const matrix& other)
{
	*this = other;
}

matrix::matrix(int _height, int _width, int _random)
{
	height = _height, width = _width;
	ents   = new MAT_ENTRY_T[_height*_width];

	// fill the matrix with random numbers
	if(_random) {
		for(int i = 0;i < _height;i++)
			for(int j = 0;j < _width;j++)
				(*this)(i, j) = rand(1.0);
	}
}

matrix::~matrix()
{
	if(size() <= 0) return ;
	delete [] ents;
}

int matrix::size() const 
{
	return height * width;
}

void matrix::print() const 
{
	if(size() <= 0) return ;
	for(int i = 0;i < height;i++) {
		for(int j = 0;j < width;j++) 
			printf("%lf ", (*this)(i, j));
		printf("\n");
	}
	printf("\n");
}

MAT_ENTRY_T& matrix::operator () (const int& y, const int& x) const 
{
	return ents[y*width+x];
}

bool matrix::operator == (const matrix& other) const 
{
	if(height != other.height || width != other.width) {
		printf("Sizes of the matrcies don't match.\n");
		return false;
	}
	
	for(int i = 0;i < height;i++)
		for(int j = 0;j < width;j++) {
			double err = abs((*this)(i, j)-other(i, j));
			if(err >= EPS) {
				printf("Check failed at (%d, %d): diff: %lf\n", i, j, err);
				return false;
			}
		}
	return true;
}

// a deep copy method
matrix& matrix::operator = (const matrix& other)
{
	if(!height && !width) {
		this->~matrix();
	}

	height = other.height;
	width  = other.width;

	ents = new MAT_ENTRY_T[height*width];

	for(int i = 0;i < height;i++)
		for(int j = 0;j < width;j++)
			(*this)(i, j) = other(i, j);

	return *this;
}

matrix& matrix::operator += (const matrix& other)
{
	if(height != other.height || width != other.width)
		return *this;

	int i;
	double t = omp_get_wtime();
#pragma omp parallel for
	for(i = 0;i < height;i++)
		for(int j = 0;j < width;j++)
			(*this)(i, j) += other(i, j);
	t = omp_get_wtime() - t;
	add_to_timer(0, 1000*t);

	return *this;
}

void matrix_add(	  matrix& c,
				const matrix& a,
				const matrix& b,
				add_type mode) 
{
	c.~matrix();
	
	const int height = a.height, width = a.width;
	if(a.height != b.height || a.width != b.width) {
		c.height = c.width = 0;
		return ;
	} else if(mode != cpu_add) {
		c.height = height, c.width = width;
		c.ents = new MAT_ENTRY_T[height*width];
	}

	if(mode == cpu_add) {
		c = a;
		c += b;
	} else {
		int size = c.size();
		MAT_ENTRY_T *d_A, *d_B, *d_C;
		gpuCheck( hipMalloc(&d_C, size*sizeof(MAT_ENTRY_T)) );
		gpuCheck( hipMalloc(&d_A, size*sizeof(MAT_ENTRY_T)) );
		gpuCheck( hipMalloc(&d_B, size*sizeof(MAT_ENTRY_T)) );

		hipEvent_t start, end;
		gpuCheck( hipEventCreate(&start) );
		gpuCheck( hipEventCreate(&end) );

		gpuCheck( hipMemcpy(d_A, a.ents, size*sizeof(MAT_ENTRY_T), hipMemcpyHostToDevice) );
		gpuCheck( hipMemcpy(d_B, b.ents, size*sizeof(MAT_ENTRY_T), hipMemcpyHostToDevice) );
		
		if(mode == gpu_add_1D) {
			constexpr int blockSize = BLOCKSIZE*BLOCKSIZE*WORKLOAD*WORKLOAD;
			dim3 blockDim(BLOCKSIZE*BLOCKSIZE),
				 gridDim((height*width+blockSize-1)/blockSize);

			gpuCheck( hipEventRecord(start) );
			add_kernel_1D<WORKLOAD*WORKLOAD><<<gridDim, blockDim>>>(d_C, d_A, d_B, height, width);
			gpuCheck( hipEventRecord(end) );
		} else {
			constexpr int blockSize = BLOCKSIZE*WORKLOAD;
			dim3 blockDim(BLOCKSIZE, BLOCKSIZE),
				 gridDim((height+blockSize-1)/blockSize, (width+blockSize-1)/blockSize);

			gpuCheck( hipEventRecord(start) );
			add_kernel_2D<WORKLOAD><<<gridDim, blockDim>>>(d_C, d_A, d_B, height, width);
			gpuCheck( hipEventRecord(end) );
		}
		
		gpuCheck( hipEventSynchronize(end) );
		gpuCheck( hipMemcpy(c.ents, d_C, size*sizeof(MAT_ENTRY_T), hipMemcpyDeviceToHost) );

		float duration = 0;
		gpuCheck( hipEventElapsedTime(&duration, start, end) );
		add_to_timer(mode, duration);

		gpuCheck( hipEventDestroy(start) );
		gpuCheck( hipEventDestroy(end) );

		gpuCheck( hipFree(d_C) );
		gpuCheck( hipFree(d_A) );
		gpuCheck( hipFree(d_B) );
	} 

	return ;
}

template<int workload>
__global__ void add_kernel_1D(MAT_ENTRY_T* d_C,
							  MAT_ENTRY_T* d_A,
							  MAT_ENTRY_T* d_B,
							  const int height, const int width)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x * workload;
	
	for(int i = 0;i < workload;i++) {
		int x = idx + blockDim.x*i;
		if(x < height*width) {
			d_C[x] = d_A[x] + d_B[x];
		}
	}
}

template<int workload>
__global__ void add_kernel_2D(MAT_ENTRY_T* d_C, 
						   	  MAT_ENTRY_T* d_A,
						   	  MAT_ENTRY_T* d_B, 
						   	  const int height, const int width)
{
	const int rIdx = threadIdx.x + blockIdx.x * blockDim.x * workload,
			  cIdx = threadIdx.y + blockIdx.y * blockDim.y * workload;

	for(int i = 0;i < workload*workload;i++) {
		int x = rIdx + blockDim.x*(i/workload),
			y = cIdx + blockDim.y*(i%workload),
			idx = x*width + y;
		if(x < height && y < width) {
			d_C[idx] = d_A[idx] + d_B[idx];
		}
	}

	// for(int i = 0;i < workload;i++) {
	// 	for(int j = 0;j < workload;j++) {
	// 		int x = rIdx + blockDim.x*i,
	// 			y = cIdx + blockDim.y*j,
	// 			idx = x*width + y;
	// 		if(x < height && y < width) {
	// 			d_C[idx] = d_A[idx] + d_B[idx];
	// 		}
	// 	}
	// }
}
