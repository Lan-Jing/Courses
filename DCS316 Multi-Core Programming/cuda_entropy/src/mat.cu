#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <ctime>
#include <cmath>
#include "../include/mat.hpp"

void init_rand() 
{
    srand(time(0));
}

char rand_char() 
{
    return (char)(rand() % 16);
}

int divup(int a, int b)
{
    return (a+b-1)/b;
}

timer recorder(10);
timer::timer(int _l)
{
    if(_l <= 0) return ;
    l = _l;
    timeArr = (double*)malloc(sizeof(double)*_l);
    init_time();
}

timer::~timer()
{
    free(timeArr);
}

void timer::init_time()
{
    for(int i = 0;i < l;i++) timeArr[i] = 0x3f3f3f3f;
}

void timer::update_time(int idx, double t)
{
    if(idx < 0 || idx >= l) return ;
    timeArr[idx] = min(timeArr[idx], t);
}

double timer::fetch_time(int idx)
{
    if(idx < 0 || idx >= l) return -1;
    return timeArr[idx];
}

mat::mat(int _m, int _n, int _mode) 
{
    m = _m, n = _n;
    val = (char*)malloc(sizeof(char)*size());
    if(_mode) {
        for(int i = 0;i < size();i++) {
            val[i] = rand_char();
        }
    } else {
        memset(val, 0, sizeof(char)*size());
    }
}

mat::~mat()
{
    free(val);
}

int mat::size() const 
{
    return m * n;
}

void mat::print() const 
{
    for(int i = 0;i < size();i++) {
        printf("%2d ", val[i]);
        if(i % n == n-1)
            printf("\n");
    }
    printf("\n");
}

float* mat::entropy_omp() 
{
    float* res = (float*)malloc(sizeof(float)*size());
    double t = omp_get_wtime();

    int i;
#pragma omp parallel for
    for(i = 0;i < m;i++) {
        for(int j = 0;j < n;j++) {
            int count[16], eleCount = 0;
            memset(count, 0, sizeof(int)*16);

            for(int dy = -2;dy <= 2;dy++) {
                for(int dx = -2;dx <= 2;dx++) {
                    int y = i + dy,
                        x = j + dx;

                    if(y >= 0 && x >= 0 && y < m && x < n) {
                        count[val[y*n+x]]++;
                        eleCount++;
                    }
                }
            }

            float entropy = 0;
            for(int k = 0;k <= 15;k++) {
                double p = (double)count[k]/eleCount;
                entropy -= p*log2(p);
            }
            res[i*n+j] = entropy;
        }
    }

    t = omp_get_wtime() - t;
    recorder.update_time(0, 1000*t);
    return res;
}

__constant__ float cd_logTable[30];
float* mat::entropy_cu(int mode)
{
    float* res = (float*)malloc(sizeof(float)*size());
    float logTable[30], logTable_1[30];
    for(int i = 0;i <= 25;i++) logTable[i]   = log2(i);
    for(int i = 0;i <= 25;i++) logTable_1[i] = i <= 1 ? 0 : i*log2(i);
    
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    char* d_val;
    float* d_entropy, *d_logTable, *d_logTable_1;
    gpuCheck( hipMalloc(&d_val, sizeof(char)*size()) );
    gpuCheck( hipMalloc(&d_entropy, sizeof(float)*size()) );
    gpuCheck( hipMalloc(&d_logTable,   sizeof(float)*30) );
    gpuCheck( hipMalloc(&d_logTable_1, sizeof(float)*30) );

    gpuCheck( hipMemcpy(d_val, val, sizeof(char)*size(), hipMemcpyHostToDevice) );
    gpuCheck( hipMemcpy(d_logTable,   logTable,   sizeof(float)*26, hipMemcpyHostToDevice) );
    gpuCheck( hipMemcpy(d_logTable_1, logTable_1, sizeof(float)*26, hipMemcpyHostToDevice) );
    gpuCheck( hipMemcpyToSymbol(HIP_SYMBOL(cd_logTable), logTable_1, sizeof(float)*26) );

    dim3 gridDim(divup(m, BLOCKSIZE), divup(n, BLOCKSIZE)),
         blockDim(BLOCKSIZE, BLOCKSIZE);

    hipEventRecord(start);
    switch(mode) {
        case 1: {
            entropy_kernel1<<<gridDim, blockDim>>>(d_entropy, d_val, m, n);
            break;
        }
        case 2: {  
            entropy_kernel2<<<gridDim, blockDim>>>(d_entropy, d_val, d_logTable, m, n);
            break;
        }
        case 3: {
            gridDim = dim3(divup(m, BLOCKSIZE-4), divup(n, BLOCKSIZE-4));
            entropy_kernel3<BLOCKSIZE-4><<<gridDim, blockDim>>>(d_entropy, d_val, m, n);
			break;
		}
        case 4: {
            gridDim = dim3(divup(m, BLOCKSIZE-4), divup(n, BLOCKSIZE-4));
            entropy_kernel4<BLOCKSIZE-4><<<gridDim, blockDim>>>(d_entropy, d_val, m, n);
			break;
		}
		case 5: {
            gridDim = dim3(divup(m, BLOCKSIZE/2), divup(n, BLOCKSIZE/2)),
            blockDim = dim3(BLOCKSIZE/2, BLOCKSIZE/2);
            entropy_kernel5<BLOCKSIZE/2, BLOCKSIZE/2><<<gridDim, blockDim>>>(d_entropy, d_val, d_logTable_1, m, n);
			break;
		}
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    gpuCheck( hipMemcpy(res, d_entropy, sizeof(float)*size(), hipMemcpyDeviceToHost) );
    gpuCheck( hipFree(d_val) );
    gpuCheck( hipFree(d_entropy) );
    gpuCheck( hipFree(d_logTable) );
    
    float t = 0;
    hipEventElapsedTime(&t, start, end);
    recorder.update_time(mode, t);

    hipEventDestroy(start);
    hipEventDestroy(end);    
    return res;
}

void print_entropy(float* entropy, int m, int n)
{
    for(int i = 0;i < m;i++) {
        for(int j = 0;j < n;j++) {
            printf("%6lf ", entropy[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}

bool check_entropy(float* entropy_a, float* entropy_b, int size)
{
    for(int i = 0;i < size*size;i++) {
        if(abs(entropy_a[i]-entropy_b[i]) > 1e-5) {
            printf("Check failed at (%d, %d): %lf, %lf\n", i/size, i%size, 
                                                           entropy_a[i], entropy_b[i]);
			return false;
        }
    }
	return true;
}

__global__ 
void entropy_kernel1(float* d_entropy, char* d_val, 
                     int m, int n)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x,
              y = threadIdx.y + blockIdx.y * blockDim.y;

    int count[16], eleCount = 0;
    memset(count, 0, sizeof(int)*16);
    for(int dx = -2;dx <= 2;dx++) {
        for(int dy = -2;dy <= 2;dy++) {
            int xx = x + dx,
                yy = y + dy;
            
            if(xx >= 0 && yy >= 0 && yy < m && xx < n) {
                count[d_val[yy*n+xx]]++;
                eleCount++;
            }
        }
    }
    eleCount = max(1, eleCount);

    float entropy = 0;
    for(int k = 0;k <= 15;k++) {
        float p = (float)count[k]/eleCount;
        entropy -= p*log2(p);
    }
    if(y < m && x < n) {
        d_entropy[y*n+x] = entropy;
    }
}

__global__ 
void entropy_kernel2(float* d_entropy, char* d_val, float* d_logTable,
                     int m, int n)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x,
              y = threadIdx.y + blockIdx.y * blockDim.y;

    char count[16], eleCount = 0;
    memset(count, 0, sizeof(char)*16);
    for(int dx = -2;dx <= 2;dx++) {
        for(int dy = -2;dy <= 2;dy++) {
            int xx = x + dx,
                yy = y + dy;
            
            if(xx >= 0 && yy >= 0 && yy < m && xx < n) {
                count[d_val[yy*n+xx]]++;
                eleCount++;
            }
        }
    }
    eleCount = max(1, eleCount);

    float entropy = 0;
    for(int k = 0;k <= 15;k++) {
        float p = (float)count[k]/eleCount;
        entropy -= p*(__ldg(&d_logTable[count[k]])-__ldg(&d_logTable[eleCount]));
	}
    if(y < m && x < n) {
        d_entropy[y*n+x] = entropy;
    }
}

template<int bsize>
__global__ void entropy_kernel3(float* d_entropy, char* d_val,
                                int m, int n)
{
    __shared__ int sd_val[32][32];
    const int x = threadIdx.x-2 + blockIdx.x * bsize,
              y = threadIdx.y-2 + blockIdx.y * bsize;
    if(x >= 0 && y >= 0 && y < m && x < n) {
        sd_val[threadIdx.y][threadIdx.x] = d_val[y*n+x];
    } else {
        sd_val[threadIdx.y][threadIdx.x] = 16;
    }
    __syncthreads();

    char count[17], eleCount = 0;
    memset(count, 0, sizeof(char)*17);

    if(threadIdx.x >= 2 && threadIdx.x <= 29 && threadIdx.y >= 2 && threadIdx.y <= 29) {
        for(int dx = -2;dx <= 2;dx++) {
            for(int dy = -2;dy <= 2;dy++) {
                char nowVal = sd_val[threadIdx.y+dy][threadIdx.x+dx];
                count[nowVal]++;
                eleCount += min(1, 16-nowVal);
            }
        }
    }

    float entropy = 0;
    for(int k = 0;k <= 15;k++) {
        float p = (float)count[k]/eleCount;
        entropy -= p*log2(p);
   }

    if(threadIdx.x >= 2 && threadIdx.x <= 29 && threadIdx.y >= 2 && threadIdx.y <= 29 && 
       y < m && x < n) {
        d_entropy[y*n+x] = entropy;
    }
}

template<int bsize>
__global__ void entropy_kernel4(float* d_entropy, char* d_val,
                                int m, int n)
{
    __shared__ int sd_val[32][32];
    const int x = threadIdx.x-2 + blockIdx.x * bsize,
              y = threadIdx.y-2 + blockIdx.y * bsize;
    if(x >= 0 && y >= 0 && y < m && x < n) {
        sd_val[threadIdx.y][threadIdx.x] = d_val[y*n+x];
    } else {
        sd_val[threadIdx.y][threadIdx.x] = 16;
    }

    char count[17], eleCount = 0;
    memset(count, 0, sizeof(char)*17);

    __syncthreads();
    if(threadIdx.x >= 2 && threadIdx.x <= 29 && threadIdx.y >= 2 && threadIdx.y <= 29) {
        for(int dx = -2;dx <= 2;dx++) {
            for(int dy = -2;dy <= 2;dy++) {
                char nowVal = sd_val[threadIdx.y+dy][threadIdx.x+dx];
                switch(nowVal) {
                    case 0:  count[0]++;  break;
                    case 1:  count[1]++;  break;
                    case 2:  count[2]++;  break;
                    case 3:  count[3]++;  break;
                    case 4:  count[4]++;  break;
                    case 5:  count[5]++;  break;
                    case 6:  count[6]++;  break;
                    case 7:  count[7]++;  break;
                    case 8:  count[8]++;  break;
                    case 9:  count[9]++;  break;
                    case 10: count[10]++; break;
                    case 11: count[11]++; break;
                    case 12: count[12]++; break;
                    case 13: count[13]++; break;
                    case 14: count[14]++; break;
                    case 15: count[15]++; break;
                    case 16: count[16]++; break;
                }
                eleCount += min(1, 16-nowVal);
            }
        }
    }

    float entropy = 0;
    for(int k = 0;k <= 15;k++) {
        float p = (float)count[k]/eleCount;
        entropy -= p*log2(p);
   }

    if(threadIdx.x >= 2 && threadIdx.x <= 29 && threadIdx.y >= 2 && threadIdx.y <= 29 && 
       y < m && x < n) {
        d_entropy[y*n+x] = entropy;
    }
}

template<int bsize_x, int bsize_y>
__global__ void entropy_kernel5(float* d_entropy, char* d_val, float* d_logTable,
                    			int m, int n)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x,
              y = threadIdx.y + blockIdx.y * blockDim.y,
			  idx = threadIdx.y*bsize_x + threadIdx.x;

    char eleCount = 0;
	// __shared__ char sd_count[16][bsize*bsize];
	// for(int i = 0;i <= 15;i++) sd_count[i][((threadIdx.y>>2)<<7)+(threadIdx.x<<2)+(threadIdx.y&3)] = 0;
    __shared__ int sd_count[16][bsize_y*bsize_x];
	for(int i = 0;i <= 15;i++) sd_count[i][idx] = 0;
    for(int dy = -2;dy <= 2;dy++) {
        for(int dx = -2;dx <= 2;dx++) {
            int xx = x + dx,
                yy = y + dy;
            
            if(xx >= 0 && yy >= 0 && yy < m && xx < n) {
				sd_count[d_val[yy*n+xx]][idx]++;
                eleCount++;
            }
        }
    }

    float entropy = 0;
    for(int k = 0;k <= 15;k++) {
		// float p = (float)sd_count[k][idx] / eleCount;
		// float p = (float)sd_count[k][idx] * (1.0 / eleCount);
        // entropy -= p*log2(p);
        // entropy -= p*(__ldg(&d_logTable[sd_count[k][idx]])-__ldg(&d_logTable[eleCount]));
		// entropy -= cd_logTable[sd_count[k][idx]];
        entropy -= __ldg(&d_logTable[sd_count[k][idx]]);
    }
    entropy = entropy / eleCount + log2(eleCount);
    if(y < m && x < n) {
        d_entropy[y*n+x] = entropy;
    }
}