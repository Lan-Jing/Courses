#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../include/conv.cuh"
#include "../include/helper.hpp"

// have the depth dimension change fastest.
int cudaImage::index(int i, int rIdx, int cIdx) const
{
    if(type == rowMajor) {
        return (rIdx * n + cIdx) * d + i;
    } else {
        return (cIdx * m + rIdx) * d + i;
    }
}

int cudaImage::get_size() const
{
    return m*n*d;
}

dtype& cudaImage::operator [] (int idx) const
{
    return mat[idx];
}

bool cudaImage::operator == (cudaImage &other) const
{
    if(m != other.m || n != other.n || d != other.d)
        return false;
    
    dtype maxError = 0;
    int rIdx;
#pragma omp parallel for private(rIdx)
    for(rIdx = 0;rIdx < m;rIdx++)
        for(int cIdx = 0;cIdx < n;cIdx++)
            for(int i = 0;i < d;i++) {
                int index = this->index(i, rIdx, cIdx);
                maxError = fmax(maxError, fabs(mat[index]-other.mat[index]));
            }
    if(maxError >= eps) {
        printf("Maximum Error: %lf, required: %lf\n", maxError, eps);
        return false;
    } else {
        return true;
    }
}

void cudaImage::operator = (cudaImage &other) 
{
    if(mat != NULL) {
        delete [] mat;
    }

    m = other.m, n = other.n, d = other.d, type = other.type;
    
    int size = get_size();
    mat = new dtype[size];

    memcpy(mat, other.mat, size*dsize);
}

cudaImage::cudaImage()
{
    m = n = d = 0;
    mat = NULL;
}

cudaImage::cudaImage(int _m, int _n, int _d, int _mode, matType _type)
{
    m = _m, n = _n, d = _d, type = _type;
    
    int size = get_size();
    mat = new dtype[size];

    if(_mode == 1) {
        for(int i = 0;i < size;i++)
        #ifdef DEBUG
            mat[i] = 1;
        #else
            mat[i] = rand_float(RAND_RBOUND);
        #endif
    }
}

cudaImage::cudaImage(cudaImage &other, matType _type)
{
    m = other.m, n = other.n, d = other.d, type = _type;

    int size = get_size();
    mat = new dtype[size];

    if(type == other.type) {
        memcpy(mat, other.mat, size*dsize);
    } else {
        for(int i = 0;i < d;i++)
            for(int rIdx = 0;rIdx < m;rIdx++)
                for(int cIdx = 0;cIdx < n;cIdx++) {
                    int index = this->index(i, rIdx, cIdx);
                    mat[index] = other.mat[index];
                }
    }
}

cudaImage::~cudaImage()
{
    delete [] mat;
}

void cudaImage::print() const
{
    printf("Arranged in %s order\n", type ? "col-major" : "row-major");
    for(int i = 0;i < d;i++) {
        printf("Channel: %d\n", i);
        for(int rIdx = 0;rIdx < m;rIdx++) {
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int index = this->index(i, rIdx, cIdx);
                printf("%lf ", mat[index]); 
            }
            printf("\n");
        }
    }
    printf("\n");
}

// Rotate the index manner as well as the matrix itself.
void cudaImage::alter_major()
{
    matType newType = type == rowMajor ? colMajor : rowMajor;
    cudaImage tmpImage = cudaImage(*this, newType);
    for(int i = 0;i < d;i++)
        for(int rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int tIdx = tmpImage.index(i, rIdx, cIdx),
                    idx = this->index(i, rIdx, cIdx);
                tmpImage[tIdx] = (*this)[idx];
            }
    *this = tmpImage;
}


// construct left and right padding for both dimensions.
// return size of the result matrix as well.
void get_padding(int &rpl, int &rpr, int &cpl, int &cpr, 
                 int &om, int &on,
                 int im, int in, int km, int kn, int s)
{
    for(int rPadding = 0;rPadding < s;rPadding++) {
        if((im-km+rPadding) % s == 0) {
            rpl = rPadding / 2;
            rpr = rPadding - rpl;
            om = (im-km+rPadding) / s + 1;
            break;
        }
    }
    for(int cPadding = 0;cPadding < s;cPadding++) {
        if((in-kn+cPadding) % s == 0) {
            cpl = cPadding / 2;
            cpr = cPadding - cpl;
            on = (in-kn+cPadding) / s + 1;
            break;
        }
    }
}


cudaImage* conv_omp(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int im = input.m, in = input.n, d = input.d,
        km = kernel.m, kn = kernel.n,
        om, on;
    int rpl, rpr, cpl, cpr;
    
    get_padding(rpl, rpr, cpl, cpr, om, on, im, in, km, kn, s);
#ifdef DEBUG
    printf("Padding Info: %d %d %d %d %d %d\n", rpl, rpr, cpl, cpr, om, on);
#endif

    resImage = new cudaImage(om, on, 1, 0, rowMajor);
    omp_stime = omp_get_wtime();

    int rIdx;
    // loop over each output pixel. BE CAREFUL to the indexing!!!
#pragma omp parallel for private(rIdx)
    for(rIdx = -rpl;rIdx < im+rpr-(km-1);rIdx += s) {
        int rIdx_o = (rIdx+rpl)/s, cIdx_o = 0;
        for(int cIdx = -cpl;cIdx < in+cpr-(kn-1);cIdx += s, cIdx_o++) {
            dtype& out = (*resImage)[resImage->index(0, rIdx_o, cIdx_o)] = 0;
            // calculate the pixel pointed by 'out'.
            for(int ky = 0;ky < km;ky++)
                for(int kx = 0;kx < kn;kx++) {
                    int iy = rIdx+ky, ix = cIdx+kx;
                    // zero padding.
                    if(iy < 0 || ix < 0 || iy >= im || ix >= in)
                        continue;
                    for(int i = 0;i < d;i++) {
                        int iIdx = input.index(i, iy, ix),
                            kIdx = kernel.index(i, ky, kx);
                        out += input[iIdx] * kernel[kIdx];
                    }
                }
        }
    }
    omp_avgTime += omp_get_wtime() - omp_stime;

    return resImage;
}

cudaImage* conv_cuda(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL, *padInput = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int im = input.m, in = input.n, d = input.d,
        km = kernel.m, kn = kernel.n,
        om, on;
    int rpl, rpr, cpl, cpr;

    get_padding(rpl, rpr, cpl, cpr, om, on, im, in, km, kn, s);
    padInput = new cudaImage(im+rpl+rpr, in+cpl+cpr, d, 0, rowMajor);
    resImage = new cudaImage(om, on, 1, 0, rowMajor);
    // set up the padded matrix
    memset(padInput->mat, 0, padInput->get_size()*dsize);
    int rIdx;
#pragma omp parallel for private(rIdx)
    for(rIdx = 0;rIdx < im;rIdx++)
        for(int cIdx = 0;cIdx < in;cIdx++) {
            int prIdx = rIdx + rpl,
                pcIdx = cIdx + cpl;
            for(int i = 0;i < d;i++) {
                int pIdx = padInput->index(i, prIdx, pcIdx),
                    idx = input.index(i, rIdx, cIdx);
                (*padInput)[pIdx] = input[idx];
            }
        }

#ifdef DEBUG
    printf("Padding Info: %d %d %d %d %d %d\n", rpl, rpr, cpl, cpr, om, on);
    padInput->print();
#endif

    dtype *d_i, *d_o, *d_k;
    hipMalloc(&d_i, padInput->get_size() * dsize);
    hipMalloc(&d_o, resImage->get_size() * dsize);
    hipMalloc(&d_k, kernel.get_size() * dsize);
    hipMemcpy(d_i, padInput->mat, padInput->get_size() * dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_k, kernel.mat, kernel.get_size() * dsize, hipMemcpyHostToDevice);

    dim3 blockDim(blockSize, blockSize),
         gridDim((om+blockSize-1)/blockSize, (on+blockSize-1)/blockSize);

    cuda_stime = omp_get_wtime();
    conv_kernel2<<<gridDim, blockDim>>>(d_i, d_k, d_o,
                                        im+rpl+rpr, in+cpl+cpr, s, d,
                                        km, kn,
                                        om, on);
    hipDeviceSynchronize();
    cuda_avgTime += omp_get_wtime() - cuda_stime;

    hipMemcpy(resImage->mat, d_o, resImage->get_size() * dsize, hipMemcpyDeviceToHost);
    hipFree(d_i);
    hipFree(d_o);
    hipFree(d_k);

    return resImage;
}

// This kernel assumes all matrices arranged in row-major order.
__global__ void conv_kernel1(vtype d_i, vtype d_k, vtype d_o, 
                             const int im, const int in, const int s, const int d,
                             const int km, const int kn, 
                             const int om, const int on)
{
    int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
        cIdx = threadIdx.y + blockIdx.y * blockDim.y,
        rIdx_s = rIdx * s,
        cIdx_s = cIdx * s;

    dtype res = 0;
    if(rIdx < om && cIdx < on) {
        int irIdx = rIdx_s;
        for(int ky = 0;ky < km;ky++, irIdx++) {
            int icIdx = cIdx_s;
            for(int kx = 0;kx < kn;kx++, icIdx++) {
                int kIdx = (ky * kn + kx) * d,
                    iIdx = (irIdx * in + icIdx) * d;
                for(int i = 0;i < d;i++, kIdx++, iIdx++) {
                    res += d_i[iIdx] * d_k[kIdx];
                }
            }
        }
        d_o[rIdx * on + cIdx] = res;
    }
}

__global__ void conv_kernel2(dtype *d_i, dtype *d_k, dtype *d_o, 
                             const int im, const int in, const int s, const int d,
                             const int km, const int kn,
                             const int om, const int on)
{
    int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
        cIdx = threadIdx.y + blockIdx.y * blockDim.y,
        rIdx_s = rIdx * s,
        cIdx_s = cIdx * s;
    
    __shared__ dtype sd_k[blockSize*blockSize];
    if(threadIdx.x < km && threadIdx.y < kn * d) {
        int kIdx = threadIdx.x * kn * d + threadIdx.y;
        sd_k[kIdx] = d_k[kIdx];
    }
    __syncthreads();

    dtype res = 0;
    if(rIdx < om && cIdx < on) {
        int irIdx = rIdx_s;
        for(int ky = 0;ky < km;ky++, irIdx++) {
            int icIdx = cIdx_s;
            for(int kx = 0;kx < kn;kx++, icIdx++) {
                int kIdx = (ky * kn + kx) * d,
                    iIdx = (irIdx * in + icIdx) * d;
                for(int i = 0;i < d;i++, kIdx++, iIdx++) {
                    res += d_i[iIdx] * sd_k[kIdx];
                }
            }
        }
        d_o[rIdx * on + cIdx] = res;
    }
}
