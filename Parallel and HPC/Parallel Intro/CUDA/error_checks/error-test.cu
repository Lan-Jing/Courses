#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG

int count = 0;

__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }

    // try to access host memory in device kernel
    // cannot pass the compile process: undefined in the kernel code.
    // count = 100;
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
  
    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    //    printf("%lf %lf\n", hA[i], hB[i]);
    }

    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK( hipMalloc((void**)&dA, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dB, sizeof(double)*N) );
    CUDA_CHECK( hipMalloc((void**)&dC, sizeof(double)*N) );
    CUDA_CHECK( hipMemcpy(dA, hA, sizeof(double)*N, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dB, hB, sizeof(double)*N, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dC, hC, sizeof(double)*N, hipMemcpyHostToDevice) );

    // Note the maximum size of threads in a block
    // One block with 256 threads
    dim3 grid(1, 1, 1), threads(ThreadsInBlock, 1, 1);
    // try to overflow the thread block: nothing changed ???
    // dim3 grid(1, 1, 1), threads((1 << 16), (1 << 16), (1 << 16));

    // try to dereference a pointer to device memory
    // the program crashed, most likely by accessing invalid address.
    // *dC = 100;

    //// Add the kernel call here
    vector_add<<<grid, threads>>>(dC, dA, dB, N);

    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    // hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    CUDA_CHECK( hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost) );
    CUDA_CHECK( hipFree(dA) );
    CUDA_CHECK( hipFree(dB) );
    CUDA_CHECK( hipFree(dC) );    

    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}