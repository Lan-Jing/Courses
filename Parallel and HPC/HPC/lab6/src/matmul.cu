#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/matmul.cuh"
#include "../include/helper.hpp"

// the cuda matricies are stored in the column major order.
// and are unfolded into 1-dimensional arrays
cudaMat* init_mat(int m, int n, int mode, matType matType) 
{
    if(m < 0 || n < 0)
        return NULL;
    
    cudaMat *mat = (cudaMat*)malloc(sizeof(cudaMat));
    mat->mat = (vtype)malloc(dsize * m * n);
    mat->m = m, mat->n = n, mat->type = matType;
    if(mode != 0) {
        for(int i = 0;i < m * n;i++)
            mat->mat[i] = rand_float(RAND_RBOUND);
    }
    return mat;
}

void free_mat(cudaMat *mat)
{
    if(mat == NULL)
        return ;
    
    free(mat->mat);
    free(mat);
}

void print_mat(cudaMat *mat)
{
    if(mat == NULL)
        return ;
    
    if(mat->type == rowMajor) {
        for(int rIdx = 0;rIdx < mat->m;rIdx++) {
            for(int cIdx = 0;cIdx < mat->n;cIdx++)
                printf("%lf ", mat->mat[rIdx * mat->n + cIdx]);
            printf("\n");
        }
    } else {
        for(int cIdx = 0;cIdx < mat->n;cIdx++) {
            for(int rIdx = 0;rIdx < mat->m;rIdx++)
                printf("%lf ", mat->mat[cIdx * mat->m + rIdx]);
            printf("\n");
        }
    }
    printf("\n");
}

// This is an extremely cache-unfriendly function, which causes frequent cache miss anyway
cudaMat* alter_majorOrder(cudaMat *mat)
{
    if(mat == NULL)
        return NULL;

    cudaMat *resMat = init_mat(mat->m, mat->n, 0, mat->type == rowMajor ? colMajor : rowMajor);
    if(mat->type == rowMajor) {
        int rIdx;
    #pragma omp parallel for private(rIdx)
        for(rIdx = 0;rIdx < mat->m;rIdx++)
            for(int cIdx = 0;cIdx < mat->n;cIdx++)
                resMat->mat[cIdx * mat->m + rIdx] = mat->mat[rIdx * mat->n + cIdx];
    } else {
        int cIdx;
    #pragma omp parallel for private(cIdx)
        for(cIdx = 0;cIdx < mat->n;cIdx++)
            for(int rIdx = 0;rIdx < mat->m;rIdx++)
                resMat->mat[rIdx * mat->n + cIdx] = mat->mat[cIdx * mat->m + rIdx];
    }

    return resMat;
}

int equal_mat(cudaMat *mata, cudaMat *matb)
{
    if(mata == NULL || matb == NULL || mata->m != matb->m || mata->n != matb->n) 
        return false;
    
    // change order if they're different
    cudaMat *matac = mata;
    if(mata->type != matb->type)
        matac = alter_majorOrder(mata);
    for(int index = 0;index < mata->m * mata->n;index++)
        if(fabs(matac->mat[index]-matb->mat[index]) > eps) {
            printf("Unsatisfied Error: %lf\n", fabs(matac->mat[index]-matb->mat[index]));
            return false;
        }

    if(matac != mata)
        free_mat(matac);
    return true;
}

cudaMat* matmul_omp(cudaMat *mata, cudaMat *matb)
{
    if(mata->n != matb->m)
        return NULL;

    cudaMat *matc = init_mat(mata->m, matb->n, 0, rowMajor);
    cudaMat *mataT = alter_majorOrder(mata);

    double stime = omp_get_wtime();
    int i;
#pragma omp parallel for private(i)
    for(i = 0;i < mata->m;i++)
        for(int j = 0;j < matb->n;j++) {
            float res = 0.0;
            for(int k = 0;k < matb->m;k++)
                res += mataT->mat[i * mata->n + k] * matb->mat[j * matb->m + k];
            matc->mat[i * mata->n + j] = res;    
        }
    printf("Execution time on %d threads CPU: %lf\n", omp_get_num_procs(), omp_get_wtime()-stime);

    free_mat(mataT);
    return matc;
}

cudaMat* matmul_cuda(cudaMat *mata, cudaMat *matb, int blockSize)
{
    if(mata->n != matb->m)
        return NULL;
    
    cudaMat *matc = init_mat(mata->m, matb->n, 0, colMajor);
    cudaMat *matbT = alter_majorOrder(matb);

    vtype d_A, d_B, d_C;
    hipMalloc(&d_A, mata->m*mata->n * dsize);
    hipMalloc(&d_B, matb->m*matb->n * dsize);
    hipMalloc(&d_C, matc->m*matc->n * dsize);

    double stime = omp_get_wtime();
    hipMemcpy(d_A, mata->mat, mata->m*mata->n * dsize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, matbT->mat, matb->m*matb->n * dsize, hipMemcpyHostToDevice);

    dim3 gridDim((matc->m+blockSize-1)/blockSize, (matc->n+blockSize-1)/blockSize), 
         blockDim(blockSize, blockSize);

    matmul_kernel1<<<gridDim, blockDim>>>(d_A, d_B, d_C, mata->m, mata->n, matb->n);

    hipDeviceSynchronize();
    hipMemcpy(matc->mat, d_C, matc->m*matc->n * dsize, hipMemcpyDeviceToHost);
    printf("Execution time on GPU: %lf\n", omp_get_wtime()-stime);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free_mat(matbT);
    return matc;
}

cudaMat* matmul_cuda_multiDevices(cudaMat *mata, cudaMat *matb, int blockSize, int num_threads)
{
    if(mata->n != matb->m)
        return NULL;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(num_threads < deviceCount) {
        printf("Exceeding available devices, maximum: %d\n", deviceCount);
        return NULL;
    }
    
    cudaMat *matc = init_mat(mata->m, matb->n, 0, colMajor);

    double stime = omp_get_wtime();
    int i;
#pragma omp parallel for private(i)
    for(i = 0;i < num_threads;i++) {
        hipSetDevice(i);
        int begin = row_begin(i, num_threads, matb->n),
            end = row_end(i, num_threads, matb->n),
            subSize = end-begin;

        vtype d_A, d_B, d_C;
        hipMalloc(&d_A, mata->m*mata->n * dsize);
        hipMalloc(&d_B, matb->m*subSize * dsize);
        hipMalloc(&d_C, matc->m*subSize * dsize);

        hipMemcpy(d_A, mata->mat, mata->m*mata->n * dsize, hipMemcpyHostToDevice);
        hipMemcpy(d_B, matb->mat + matb->m*begin, matb->m*subSize * dsize, hipMemcpyHostToDevice);

        dim3 gridDim((matc->m+blockSize-1)/blockSize, (subSize+blockSize-1)/blockSize), 
             blockDim(blockSize, blockSize);

        matmul_kernel2<<<gridDim, blockDim>>>(d_A, d_B, d_C, mata->m , mata->n, subSize);

        hipMemcpy(matc->mat + matc->m*begin, d_C, matc->m*subSize * dsize, hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    printf("Execution time on %d GPUs: %lf\n", num_threads, omp_get_wtime()-stime);

    free_mat(mata);
    free_mat(matb);
    return matc;
}

// Matrix A in column major order, B in row major
__global__ void matmul_kernel1(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
    const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
              cIdx = threadIdx.y + blockIdx.y * blockDim.y,
              index = cIdx * m + rIdx;    
    if(rIdx < m && cIdx < k) {
        float res = 0;
        for(int i = 0;i < n;i++)
            res += d_A[i * m + rIdx] * d_B[i * k + cIdx];
        d_C[index] = res;
    }
}

// Matrix B shoule be in column major order.
__global__ void matmul_kernel2(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
    const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
              cIdx = threadIdx.y + blockIdx.y * blockDim.y,
              index = cIdx * m + rIdx;    
    if(rIdx < m && cIdx < k) {
        float res = 0;
        for(int i = 0;i < n;i++)
            res += d_A[i * m + rIdx] * d_B[cIdx * n + i];
        d_C[index] = res;
    }
}