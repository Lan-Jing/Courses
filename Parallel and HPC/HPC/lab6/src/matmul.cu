#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../include/matmul.cuh"
#include "../include/helper.hpp"

// the cuda matricies are stored in the column major order.
// and are unfolded into 1-dimensional arrays
cudaMat* init_mat(int m, int n, int mode, matType matType) 
{
	if(m < 0 || n < 0)
		return NULL;
	
	cudaMat *mat = (cudaMat*)malloc(sizeof(cudaMat));
	mat->mat = (vtype)malloc(dsize * m * n);
	mat->m = m, mat->n = n, mat->type = matType;
	if(mode != 0) {
		for(int i = 0;i < m * n;i++)
			mat->mat[i] = rand_float(RAND_RBOUND);
	}
	return mat;
}

void free_mat(cudaMat *mat)
{
	if(mat == NULL)
		return ;
	
	free(mat->mat);
	free(mat);
}

void print_mat(cudaMat *mat)
{
	if(mat == NULL)
		return ;
	
	if(mat->type == rowMajor) {
		for(int rIdx = 0;rIdx < mat->m;rIdx++) {
			for(int cIdx = 0;cIdx < mat->n;cIdx++)
				printf("%lf ", mat->mat[rIdx * mat->n + cIdx]);
			printf("\n");
		}
	} else {
		for(int cIdx = 0;cIdx < mat->n;cIdx++) {
			for(int rIdx = 0;rIdx < mat->m;rIdx++)
				printf("%lf ", mat->mat[cIdx * mat->m + rIdx]);
			printf("\n");
		}
	}
	printf("\n");
}

// This is an extremely cache-unfriendly function, which causes frequent cache miss anyway
cudaMat* alter_majorOrder(cudaMat *mat)
{
	if(mat == NULL)
		return NULL;

	cudaMat *resMat = init_mat(mat->m, mat->n, 0, mat->type == rowMajor ? colMajor : rowMajor);
	if(mat->type == rowMajor) {
		int rIdx;
	#pragma omp parallel for private(rIdx)
		for(rIdx = 0;rIdx < mat->m;rIdx++)
			for(int cIdx = 0;cIdx < mat->n;cIdx++)
				resMat->mat[cIdx * mat->m + rIdx] = mat->mat[rIdx * mat->n + cIdx];
	} else {
		int cIdx;
	#pragma omp parallel for private(cIdx)
		for(cIdx = 0;cIdx < mat->n;cIdx++)
			for(int rIdx = 0;rIdx < mat->m;rIdx++)
				resMat->mat[rIdx * mat->n + cIdx] = mat->mat[cIdx * mat->m + rIdx];
	}

	return resMat;
}

int equal_mat(cudaMat *mata, cudaMat *matb)
{
	if(mata == NULL || matb == NULL || mata->m != matb->m || mata->n != matb->n) 
		return false;
	
	// change order if they're different
	cudaMat *matac = mata->type != matb->type ? alter_majorOrder(mata) : mata;

	double maxError = 0;
	for(int index = 0;index < mata->m * mata->n;index++)
		maxError = fmax(maxError, fabs(matac->mat[index]-matb->mat[index])/matb->mat[index]);

	if(matac != mata)
		free_mat(matac);

	printf("Maximum error: %lf, required: %lf\n", maxError, eps);
	return maxError > eps ? false : true;
}

cudaMat* matmul_omp(cudaMat *mata, cudaMat *matb)
{
	if(mata->n != matb->m)
		return NULL;

	cudaMat *matc = init_mat(mata->m, matb->n, 0, rowMajor);
	cudaMat *mataT = alter_majorOrder(mata);

	double stime = omp_get_wtime(), etime;
	int i;
#pragma omp parallel for private(i)
	for(i = 0;i < mata->m;i++)
		for(int j = 0;j < matb->n;j++) {
			float res = 0.0;
			for(int k = 0;k < matb->m;k++)
				res += mataT->mat[i * mata->n + k] * matb->mat[j * matb->m + k];
			matc->mat[i * mata->n + j] = res;    
		}
	etime = omp_get_wtime();
	avgTime_omp += etime-stime;

	free_mat(mataT);
	return matc;
}

cudaMat* matmul_cuda(cudaMat *mata, cudaMat *matb)
{
	if(mata->n != matb->m)
		return NULL;

	cudaMat *matc = init_mat(mata->m, matb->n, 0, colMajor);
	cudaMat *matbT = alter_majorOrder(matb);

	vtype d_A, d_B, d_C;
	hipMalloc(&d_A, mata->m*mata->n * dsize);
	hipMalloc(&d_B, matb->m*matb->n * dsize);
	hipMalloc(&d_C, matc->m*matc->n * dsize);
	hipMemcpy(d_A, mata->mat, mata->m*mata->n * dsize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, matbT->mat, matb->m*matb->n * dsize, hipMemcpyHostToDevice);
	
	dim3 gridDim((matc->m+blockSize-1)/blockSize, (matc->n+blockSize-1)/blockSize), 
		 blockDim(blockSize, subBlockSize);

	double stime = omp_get_wtime();
	matmul_kernel5<<<gridDim, blockDim>>>(d_A, d_B, d_C, mata->m, mata->n, matb->n);
	hipDeviceSynchronize();
	avgTime_cuda += omp_get_wtime()-stime;

	hipMemcpy(matc->mat, d_C, matc->m*matc->n * dsize, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free_mat(matbT);
	return matc;
}

cudaMat* matmul_cuda_multiDevices(cudaMat *mata, cudaMat *matb, int num_threads)
{
	if(mata->n != matb->m)
		return NULL;
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if(num_threads < deviceCount) {
		printf("Exceeding available devices, maximum: %d\n", deviceCount);
		return NULL;
	}
	
	cudaMat *matc = init_mat(mata->m, matb->n, 0, colMajor);

	double stime;
	int i;
#pragma omp parallel for private(i)
	for(i = 0;i < num_threads;i++) {
		hipSetDevice(i);
		int begin = row_begin(i, num_threads, matb->n),
			end = row_end(i, num_threads, matb->n),
			subSize = end-begin;

		vtype d_A, d_B, d_C;
		hipMalloc(&d_A, mata->m*mata->n * dsize);
		hipMalloc(&d_B, matb->m*subSize * dsize);
		hipMalloc(&d_C, matc->m*subSize * dsize);

		hipMemcpy(d_A, mata->mat, mata->m*mata->n * dsize, hipMemcpyHostToDevice);
		hipMemcpy(d_B, matb->mat + matb->m*begin, matb->m*subSize * dsize, hipMemcpyHostToDevice);

		dim3 gridDim((matc->m+blockSize-1)/blockSize, (subSize+blockSize-1)/blockSize), 
			 blockDim(blockSize, blockSize);

		#pragma omp barrior
		if(i == 0) stime = omp_get_wtime();

		matmul_kernel4<<<gridDim, blockDim>>>(d_A, d_B, d_C, mata->m , mata->n, subSize);
		hipDeviceSynchronize();
		
		#pragma omp barrior
		if(i == 0) avgTime_ompCuda += omp_get_wtime()-stime;

		hipMemcpy(matc->mat + matc->m*begin, d_C, matc->m*subSize * dsize, hipMemcpyDeviceToHost);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
	}
	return matc;
}

cudaMat* matmul_cuBLAS(cudaMat *mata, cudaMat *matb, int num_threads)
{
	if(mata->n != matb->m)
		return NULL;
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if(num_threads > deviceCount) {
		printf("Exceeding available devices, maximum: %d\n", deviceCount);
		return NULL;
	}
	
	cudaMat *matc = init_mat(mata->m, matb->n, 0, colMajor);

	double stime;
	int i;
#pragma omp parallel for private(i)
	for(i = 0;i < num_threads;i++) {
		hipblasHandle_t handle_i;
		hipSetDevice(i);
		hipblasCreate(&handle_i);

		int begin = row_begin(i, num_threads, matb->n),
			end = row_end(i, num_threads, matb->n),
			subSize = end-begin;

		vtype d_A, d_B, d_C;
		hipMalloc(&d_A, mata->m*mata->n * dsize);
		hipMalloc(&d_B, matb->m*subSize * dsize);
		hipMalloc(&d_C, matc->m*subSize * dsize);

		hipMemcpy(d_A, mata->mat, mata->m*mata->n * dsize, hipMemcpyHostToDevice);
		hipMemcpy(d_B, matb->mat + matb->m*begin, matb->m*subSize * dsize, hipMemcpyHostToDevice);

		#pragma omp barrior
		if(i == 0) stime = omp_get_wtime();

		dtype alpha = 1.0, beta = 0.0;
		hipblasDgemm(handle_i, 
					HIPBLAS_OP_N, 
					HIPBLAS_OP_N, 
					mata->m, subSize, mata->n, 
					&alpha, 
					d_A, mata->m,
					d_B, matb->m,
					&beta, 
					d_C, matc->m);

		hipDeviceSynchronize();
		#pragma omp barrior
		if(i == 0) avgTime_cublas += omp_get_wtime()-stime;

		hipMemcpy(matc->mat + matc->m*begin, d_C, matc->m*subSize * dsize, hipMemcpyDeviceToHost);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		hipblasDestroy(handle_i);
	}
	return matc;
}

// Matrix A in column-major order, B in row-major
__global__ void matmul_kernel1(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
	const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
			  cIdx = threadIdx.y + blockIdx.y * blockDim.y,
			  index = cIdx * m + rIdx;
	dtype res = 0;
	for(int i = 0;i < n;i++)
		res += d_A[i * m + rIdx] * d_B[i * k + cIdx];
	if(rIdx < m && cIdx < k) 
		d_C[index] = res;
}

// In this kernel A and B should both be in column-major order
__global__ void matmul_kernel2(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
	const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
			  cIdx = threadIdx.y + blockIdx.y * blockDim.y,
			  index = cIdx * m + rIdx;    
	dtype res = 0;
	for(int i = 0;i < n;i++)
		res += d_A[i * m + rIdx] * d_B[cIdx * n + i];
	if(rIdx < m && cIdx < k) 
		d_C[index] = res;
}

// Using shared memory based on kernel 1.
// Note that shared memory is managed in a row-major order.
__global__ void matmul_kernel3(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
	const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
			  cIdx = threadIdx.y + blockIdx.y * blockDim.y,
			  index = cIdx * m + rIdx;

	dtype res = 0;
	__shared__ dtype sd_A[blockSize+1][blockSize+1], 
					 sd_B[blockSize+1][blockSize+1];
	for(int bIdx = 0;bIdx < n;bIdx += blockSize) {
		sd_A[threadIdx.y][threadIdx.x] = (rIdx < m && bIdx+threadIdx.y < n) 
									   ? d_A[(bIdx+threadIdx.y) * m + rIdx]
									   : 0;
		sd_B[threadIdx.y][threadIdx.x] = (cIdx < k && bIdx+threadIdx.x < n) 
									   ? d_B[(bIdx+threadIdx.x) * k + cIdx]
									   : 0;
		__syncthreads();
		for(int i = 0;i < blockSize;i++)
			res += sd_A[i][threadIdx.x] * sd_B[threadIdx.y][i];
		__syncthreads();
	}
	if(rIdx < m && cIdx < k) 
		d_C[index] = res;
}

// Based on kernel 2, assuming matrix B in row-major order.
__global__ void matmul_kernel4(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
	const int rIdx = threadIdx.x + blockIdx.x * blockDim.x,
			  cIdx = threadIdx.y + blockIdx.y * blockDim.y,
			  index = cIdx * m + rIdx;    

	dtype res = 0;
	__shared__ dtype sd_A[blockSize+1][blockSize+1], 
					 sd_B[blockSize+1][blockSize+1];
	for(int bIdx = 0;bIdx < n;bIdx += blockSize) {
		sd_A[threadIdx.y][threadIdx.x] = (rIdx < m && bIdx+threadIdx.y < n) 
									   ? d_A[(bIdx+threadIdx.y) * m + rIdx]
									   : 0;
		sd_B[threadIdx.y][threadIdx.x] = (cIdx < k && bIdx+threadIdx.x < n) 
									   ? d_B[cIdx * n + bIdx+threadIdx.x]
									   : 0;
		__syncthreads();
		for(int i = 0;i < blockSize;i++)
			res += sd_A[i][threadIdx.x] * sd_B[threadIdx.y][i];
		__syncthreads();
	}
	if(rIdx < m && cIdx < k) 
		d_C[index] = res;
}

// more work per thread.
__global__ void matmul_kernel5(vtype d_A, vtype d_B, vtype d_C, int m, int n, int k)
{
	const int b_rIdx = threadIdx.x, 
			  b_cIdx = threadIdx.y,
			  rIdx = b_rIdx + blockIdx.x * blockSize,
			  cIdx = b_cIdx + blockIdx.y * blockSize;
	
	dtype res[elementPerBlock];
	memset(res, 0, elementPerBlock*dsize);
	__shared__ dtype sd_A[blockSize+1][blockSize+1],
					 sd_B[blockSize+1][blockSize+1];

	for(int bIdx = 0;bIdx < n;bIdx += blockSize) {
		const int t_rIdx = bIdx + b_rIdx,
				  t_cIdx = bIdx + b_cIdx;
		for(int i = 0;i < elementPerBlock;i++) {
			sd_A[b_cIdx+i*subBlockSize][b_rIdx] = (rIdx < m && t_cIdx+i*subBlockSize < n)
												? d_A[(t_cIdx+i*subBlockSize) * m + rIdx]
												: 0;
			sd_B[b_rIdx][b_cIdx+i*subBlockSize] = (cIdx+i*subBlockSize < k && t_rIdx < n)
												? d_B[t_rIdx * k + cIdx+i*subBlockSize]
												: 0;
		}
		__syncthreads();
		for(int j = 0;j < blockSize;j++)
			for(int i = 0;i < elementPerBlock;i++) 
				res[i] += sd_A[j][b_rIdx] * sd_B[j][b_cIdx+i*subBlockSize];
		__syncthreads();
	}
	for(int i = 0;i < elementPerBlock;i++) {
		int index = (cIdx+i*subBlockSize) * m + rIdx;
		if(rIdx < m && cIdx+i*subBlockSize < k)
			d_C[index] = res[i];
	}
}