#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>

#include "../include/matmul.cuh"
#include "../include/helper.hpp"

#define blockSize 32
#define testRounds 1
// #define DEBUG

int main(int argc, char *argv[])
{
    init_rand();

    int m, n, k;
    m = n = k = strtol(argv[1], NULL, 10);
    printf("Conducting Multiplication to two matricies with size %d*%d\n", m, n);
    cudaMat *mata = init_mat(m, n, 1, colMajor),
            *matb = init_mat(n, k, 1, colMajor);

    for(int round = 0;round < testRounds;round++) {
        printf("Testing Round %d:\n", round);

        // cudaMat *omp_matc = matmul_omp(mata, matb);
        cudaMat *cuda_matc = matmul_cuda(mata, matb, blockSize);
        cudaMat *ompCuda_matc = matmul_cuda_multiDevices(mata, matb, blockSize, 4);
    
    #ifdef DEBUG
        print_mat(omp_matc);
        print_mat(cuda_matc);
        print_mat(ompCuda_matc);
    #endif
        
        printf("Result check %s\n\n", equal_mat(ompCuda_matc, cuda_matc) ? "accepted" : "failed");
        // free_mat(omp_matc);
        free_mat(cuda_matc);
        free_mat(ompCuda_matc);
    }
    return 0;
}