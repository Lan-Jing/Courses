#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../include/helper.hpp"

void init_rand()
{
    srand(time(0));
}

// construct a float number in [0, rbound]
float rand_float(float rbound) 
{
    return (float)((double)rand()/(double)(RAND_MAX/rbound));
}

int Max(int a, int b)
{
    return a > b ? a : b;
}

int row_begin(int rank, int num_threads, int size)
{
    int blockSize = size/num_threads, margin = size%num_threads;
    return (blockSize+1)*rank - Max(rank-margin, 0);
}

int row_end(int rank, int num_threads, int size)
{
    int blockSize = size/num_threads, margin = size%num_threads;
    return row_begin(rank, num_threads, size) + blockSize + (rank < margin);
}

// check if there's error when executing the last step.
void CUDA_CHECK_LAST()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));
}