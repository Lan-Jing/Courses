#include <iostream>
using namespace std;
#include <stdio.h>
#include <math.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../include/conv.cuh"
#include "../include/helper.hpp"

int cudaImage::index(int i, int rIdx, int cIdx) const
{
    if(type == rowMajor) {
        return (rIdx * n + cIdx) * d + i;
    } else {
        return (cIdx * m + rIdx) * d + i;
    }
}

int cudaImage::get_size() const
{
    return m*n*d;
}

dtype& cudaImage::operator [] (int idx)
{
    return mat[idx];
}

bool cudaImage::operator == (cudaImage &other)
{
    if(m != other.m || n != other.n || d != other.d)
        return false;
    
    for(int i = 0;i < d;i++)
        for(int rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int index = this->index(i, rIdx, cIdx);
                if(mat[index] != other.mat[index])
                    return false;
            } 
    return true;
}

void cudaImage::operator = (cudaImage &other) 
{
    if(mat != NULL) {
        delete [] mat;
    }

    m = other.m, n = other.n, d = other.d, type = other.type;
    
    int size = get_size();
    mat = new dtype[size];

    memcpy(mat, other.mat, size*dsize);
}

cudaImage::cudaImage()
{
    m = n = d = 0;
    mat = NULL;
}

cudaImage::cudaImage(int _m, int _n, int _d, int _mode, matType _type)
{
    m = _m, n = _n, d = _d, type = _type;
    
    int size = get_size();
    mat = new dtype[size];

    if(_mode == 1) {
        for(int i = 0;i < size;i++)
        #ifdef DEBUG
            mat[i] = 1;
        #else
            mat[i] = rand_float(RAND_RBOUND);
        #endif
    }
}

cudaImage::cudaImage(cudaImage &other, matType _type)
{
    m = other.m, n = other.n, d = other.d, type = _type;

    int size = get_size();
    mat = new dtype[size];

    if(type == other.type) {
        memcpy(mat, other.mat, size*dsize);
    } else {
        for(int i = 0;i < d;i++)
            for(int rIdx = 0;rIdx < m;rIdx++)
                for(int cIdx = 0;cIdx < n;cIdx++) {
                    int index = this->index(i, rIdx, cIdx);
                    mat[index] = other.mat[index];
                }
    }
}

cudaImage::~cudaImage()
{
    delete [] mat;
}

void cudaImage::print() const
{
    printf("Arranged in %s order\n", type ? "col-major" : "row-major");
    for(int i = 0;i < d;i++) {
        printf("Channel: %d\n", i);
        for(int rIdx = 0;rIdx < m;rIdx++) {
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int index = this->index(i, rIdx, cIdx);
                printf("%lf ", mat[index]); 
            }
            printf("\n");
        }
    }
    printf("\n");
}

// Rotate the index manner as well as the matrix itself.
void cudaImage::alter_major()
{
    matType newType = type == rowMajor ? colMajor : rowMajor;
    cudaImage tmpImage = cudaImage(*this, newType);
    for(int i = 0;i < d;i++)
        for(int rIdx = 0;rIdx < m;rIdx++)
            for(int cIdx = 0;cIdx < n;cIdx++) {
                int tIdx = tmpImage.index(i, rIdx, cIdx),
                    idx = this->index(i, rIdx, cIdx);
                tmpImage[tIdx] = (*this)[idx];
            }
    *this = tmpImage;
}

cudaImage* conv_omp(cudaImage &input, cudaImage &kernel, int s)
{
    cudaImage *resImage = NULL;
    if(input.d != kernel.d || kernel.m > input.m || kernel.n > input.n)
        return resImage;
    
    int im = input.m, in = input.n, d = input.d,
        km = kernel.m, kn = kernel.n,
        om, on;
    int rpl, rpr, cpl, cpr;
    // compute left and right padding for both dimensions.
    for(int rPadding = 0;rPadding < s;rPadding++) {
        if((im-km+rPadding) % s == 0) {
            rpl = rPadding / 2;
            rpr = rPadding - rpl;
            om = (im-km+rPadding) / s + 1;
            break;
        }
    }
    for(int cPadding = 0;cPadding < s;cPadding++) {
        if((in-kn+cPadding) % s == 0) {
            cpl = cPadding / 2;
            cpr = cPadding - cpl;
            on = (in-kn+cPadding) / s + 1;
            break;
        }
    }
#ifdef DEBUG
    printf("Padding Info: %d %d %d %d %d %d\n", rpl, rpr, cpl, cpr, om, on);
#endif

    resImage = new cudaImage(om, on, 1, 0, rowMajor);
    int rIdx;
    // loop over each output pixel. BE CAREFUL to the indexing!!!
#pragma omp parallel for private(rIdx)
    for(rIdx = -rpl;rIdx < im+rpr-(km-1);rIdx += s) {
        int rIdx_o = (rIdx+rpl)/s, cIdx_o = 0;
        for(int cIdx = -cpl;cIdx < in+cpr-(kn-1);cIdx += s, cIdx_o++) {
            dtype& out = (*resImage)[resImage->index(0, rIdx_o, cIdx_o)] = 0;
            // calculate this pixel.
            for(int ky = 0;ky < km;ky++)
                for(int kx = 0;kx < kn;kx++) {
                    int iy = rIdx+ky, ix = cIdx+kx;
                    // zero padding.
                    if(iy < 0 || ix < 0 || iy >= im || ix >= in)
                        continue;
                    for(int i = 0;i < d;i++) {
                        int iIdx = input.index(i, iy, ix),
                            kIdx = kernel.index(i, ky, kx);
                        out += input[iIdx] * kernel[kIdx];
                    }
                }
        }
    }

    return resImage;
}